#include "hip/hip_runtime.h"
#include "LBM_GPU.cuh"
#include <cmath>

ofstream fout_GPU("out_GPU.dat");
ofstream fout_GPU_Ux("out_GPU_Ux.dat");
ofstream fout_GPU_Uy("out_GPU_Uy.dat");
ifstream fin_GPU("in_GPU.txt");

LBM_GPU::LBM_GPU()
{
// ============================================================================ //
//  LOAD THE PARAMETERS
// ============================================================================ //
	fin_GPU >> nx;				fin_GPU >> comment;
	fin_GPU >> ny;				fin_GPU >> comment;
	fin_GPU >> Lx;				fin_GPU >> comment;
	fin_GPU >> Ly;				fin_GPU >> comment;
	fin_GPU >> a;				fin_GPU >> comment;
	fin_GPU >> Re;				fin_GPU >> comment;
	fin_GPU >> Ux0;				fin_GPU >> comment;
	fin_GPU >> BLOCK_SIZE_X;	fin_GPU >> comment;
	fin_GPU >> BLOCK_SIZE_Y;	fin_GPU >> comment;
	fin_GPU >> BLOCK_SIZE_Z;	fin_GPU >> comment;
// ============================================================================ //

	
// ============================================================================ //
//  NEW & CUDAMALLOC
// ============================================================================ //
	is_boundary_node = new int[nx*ny];	hipMalloc((void**)&d_is_boundary_node, nx*ny * sizeof(int));
	U = new float[nx*ny];				hipMalloc((void**)&d_U, nx*ny * sizeof(float));
	Ux = new float[nx*ny];				hipMalloc((void**)&d_Ux, nx*ny * sizeof(float));
	Uy = new float[nx*ny];				hipMalloc((void**)&d_Uy, nx*ny * sizeof(float));
	rho = new float[nx*ny];				hipMalloc((void**)&d_rho, nx*ny * sizeof(float));
	W = new float[nx*ny];
	UN = new float[nx*ny];				hipMalloc((void**)&d_UN, nx*ny * sizeof(float));
	UxN = new float[nx*ny];				hipMalloc((void**)&d_UxN, nx*ny * sizeof(float));
	UyN = new float[nx*ny];				hipMalloc((void**)&d_UyN, nx*ny * sizeof(float));
	rhoN = new float[nx*ny];			hipMalloc((void**)&d_rhoN, nx*ny * sizeof(float));
	f = new float[nx*ny*a];				hipMalloc((void**)&d_f, nx*ny*a * sizeof(float));
	ftemp = new float[nx*ny*a];			hipMalloc((void**)&d_ftemp, nx*ny*a * sizeof(float));
	fN = new float[nx*ny*a];			hipMalloc((void**)&d_fN, nx*ny*a * sizeof(float));
	feq = new float[nx*ny*a];			hipMalloc((void**)&d_feq, nx*ny*a * sizeof(float));
	ex = new float[a];					hipMalloc((void**)&d_ex, a * sizeof(float));
	ey = new float[a];					hipMalloc((void**)&d_ey, a * sizeof(float));
	U_p = new float[nx*ny];
	Ux_p = new float[nx*ny];
	Uy_p = new float[nx*ny];
// ============================================================================ //


// ============================================================================ //
//  Microscopic velocity
// ============================================================================ //
	ex[0] = 0.0,	ey[0] = 0.0;
	ex[1] = 1.0,	ey[1] = 0.0;
	ex[2] = 0.0,	ey[2] = 1.0;
	ex[3] = -1.0,	ey[3] = 0.0;
	ex[4] = 0.0,	ey[4] = -1.0;
	ex[5] = 1.0,	ey[5] = 1.0;
	ex[6] = -1.0,	ey[6] = 1.0;
	ex[7] = -1.0,	ey[7] = -1.0;
	ex[8] = 1.0,	ey[8] = -1.0;
	hipMemcpy(d_ex, ex, a * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_ey, ey, a * sizeof(float), hipMemcpyHostToDevice);
// ============================================================================ //



// ============================================================================ //
//  SET BOUNDARY NODE
// ============================================================================ //
	for (i = 0; i < nx; i++) {
		for (j = 0; j < ny; j++) {
			if (i == 0 || i == nx - 1 || j == 0 || j == ny - 1) is_boundary_node[i + nx*j] = 1;
			else is_boundary_node[i + nx*j] = 0;
		}
	}
	hipMemcpy(d_is_boundary_node, is_boundary_node, nx*ny * sizeof(int), hipMemcpyHostToDevice);
// ============================================================================ //




// ============================================================================ //
//  INITIAL CONDITION
// ============================================================================ //
	del_x = Lx / (float)nx;
	del_y = Ly / (float)ny;
	del_t = pow(del_x, 2);

	Ux0_p = Ux0 * (del_x / del_t);
	tau = 3.0*(del_t / pow(del_x, 2))*(Ux0_p * Lx / Re) + 0.5;

	nu = (1.0 / 3.0)*(tau - 0.5);

	for (i = 0; i < nx; i++) {
		for (j = 0; j < ny; j++) {

			rho[i + nx*j] = 1.0;

			f[i + nx*j + nx*ny * 0] = (4.0 / 9.0) * rho[i + nx*j];
			f[i + nx*j + nx*ny * 1] = (1.0 / 9.0) * rho[i + nx*j];
			f[i + nx*j + nx*ny * 2] = (1.0 / 9.0) * rho[i + nx*j];
			f[i + nx*j + nx*ny * 3] = (1.0 / 9.0) * rho[i + nx*j];
			f[i + nx*j + nx*ny * 4] = (1.0 / 9.0) * rho[i + nx*j];
			f[i + nx*j + nx*ny * 5] = (1.0 / 36.0) * rho[i + nx*j];
			f[i + nx*j + nx*ny * 6] = (1.0 / 36.0) * rho[i + nx*j];
			f[i + nx*j + nx*ny * 7] = (1.0 / 36.0) * rho[i + nx*j];
			f[i + nx*j + nx*ny * 8] = (1.0 / 36.0) * rho[i + nx*j];
		}
	}
	hipMemcpy(d_rho, rho, nx*ny * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_f, f, nx*ny*a * sizeof(float), hipMemcpyHostToDevice);
// ============================================================================ //


}

__global__ 
void Kernel_Streaming(float* f, float* ftemp, int* is_boundary_node, int nx, int ny, int a) {

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	int k = blockDim.z * blockIdx.z + threadIdx.z;
	if (i >= nx || j >= ny || k >= a) return;

	int in, ip, jn, jp;


	if (!is_boundary_node[i + nx*j]) {

		in = i - 1;
		ip = i + 1;
		jn = j - 1;
		jp = j + 1;

		ftemp[i + nx*j + nx*ny * 0] = f[i + nx*j + nx*ny * 0];
		ftemp[ip + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 1];
		ftemp[i + nx*jp + nx*ny * 2] = f[i + nx*j + nx*ny * 2];
		ftemp[in + nx*j + nx*ny * 3] = f[i + nx*j + nx*ny * 3];
		ftemp[i + nx*jn + nx*ny * 4] = f[i + nx*j + nx*ny * 4];
		ftemp[ip + nx*jp + nx*ny * 5] = f[i + nx*j + nx*ny * 5];
		ftemp[in + nx*jp + nx*ny * 6] = f[i + nx*j + nx*ny * 6];
		ftemp[in + nx*jn + nx*ny * 7] = f[i + nx*j + nx*ny * 7];
		ftemp[ip + nx*jn + nx*ny * 8] = f[i + nx*j + nx*ny * 8];
	}
	else if ((i == 0) && (j > 0 && j < ny - 1)) {				//LEFT

		in = i - 1;
		ip = i + 1;
		jn = j - 1;
		jp = j + 1;

		ftemp[i + nx*j + nx*ny * 0] = f[i + nx*j + nx*ny * 0];
		ftemp[ip + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 1];
		ftemp[i + nx*jp + nx*ny * 2] = f[i + nx*j + nx*ny * 2];
		ftemp[i + nx*jn + nx*ny * 4] = f[i + nx*j + nx*ny * 4];
		ftemp[ip + nx*jp + nx*ny * 5] = f[i + nx*j + nx*ny * 5];
		ftemp[ip + nx*jn + nx*ny * 8] = f[i + nx*j + nx*ny * 8];
	}
	else if ((i > 0 && i < nx - 1) && (j == ny - 1)) {			//TOP

		in = i - 1;
		ip = i + 1;
		jn = j - 1;
		jp = j + 1;

		ftemp[i + nx*j + nx*ny * 0] = f[i + nx*j + nx*ny * 0];
		ftemp[ip + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 1];
		ftemp[in + nx*j + nx*ny * 3] = f[i + nx*j + nx*ny * 3];
		ftemp[i + nx*jn + nx*ny * 4] = f[i + nx*j + nx*ny * 4];
		ftemp[in + nx*jn + nx*ny * 7] = f[i + nx*j + nx*ny * 7];
		ftemp[ip + nx*jn + nx*ny * 8] = f[i + nx*j + nx*ny * 8];
	}
	else if ((i > 0 && i < nx - 1) && (j == 0)) {				//BOTTOM

		in = i - 1;
		ip = i + 1;
		jn = j - 1;
		jp = j + 1;

		ftemp[i + nx*j + nx*ny * 0] = f[i + nx*j + nx*ny * 0];
		ftemp[ip + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 1];
		ftemp[i + nx*jp + nx*ny * 2] = f[i + nx*j + nx*ny * 2];
		ftemp[in + nx*j + nx*ny * 3] = f[i + nx*j + nx*ny * 3];
		ftemp[ip + nx*jp + nx*ny * 5] = f[i + nx*j + nx*ny * 5];
		ftemp[in + nx*jp + nx*ny * 6] = f[i + nx*j + nx*ny * 6];
	}
	else if ((i == nx - 1) && (j > 0 && j < ny - 1)) {			//RIGHT

		in = i - 1;
		ip = i + 1;
		jn = j - 1;
		jp = j + 1;

		ftemp[i + nx*j + nx*ny * 0] = f[i + nx*j + nx*ny * 0];
		ftemp[i + nx*jp + nx*ny * 2] = f[i + nx*j + nx*ny * 2];
		ftemp[in + nx*j + nx*ny * 3] = f[i + nx*j + nx*ny * 3];
		ftemp[i + nx*jn + nx*ny * 4] = f[i + nx*j + nx*ny * 4];
		ftemp[in + nx*jp + nx*ny * 6] = f[i + nx*j + nx*ny * 6];
		ftemp[in + nx*jn + nx*ny * 7] = f[i + nx*j + nx*ny * 7];
	}
	else if ((i == 0) && (j == 0)) {							//BOTTOM-LEFT

		in = i - 1;
		ip = i + 1;
		jn = j - 1;
		jp = j + 1;

		ftemp[i + nx*j + nx*ny * 0] = f[i + nx*j + nx*ny * 0];
		ftemp[ip + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 1];
		ftemp[i + nx*jp + nx*ny * 2] = f[i + nx*j + nx*ny * 2];
		ftemp[ip + nx*jp + nx*ny * 5] = f[i + nx*j + nx*ny * 5];
	}
	else if ((i == 0) && (j == ny - 1)) {						//TOP-LEFT

		in = i - 1;
		ip = i + 1;
		jn = j - 1;
		jp = j + 1;

		ftemp[i + nx*j + nx*ny * 0] = f[i + nx*j + nx*ny * 0];
		ftemp[ip + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 1];
		ftemp[i + nx*jn + nx*ny * 4] = f[i + nx*j + nx*ny * 4];
		ftemp[ip + nx*jn + nx*ny * 8] = f[i + nx*j + nx*ny * 8];
	}
	else if ((i == nx - 1) && (j == ny - 1)) {					//TOP-RIGHT

		in = i - 1;
		ip = i + 1;
		jn = j - 1;
		jp = j + 1;

		ftemp[i + nx*j + nx*ny * 0] = f[i + nx*j + nx*ny * 0];
		ftemp[in + nx*j + nx*ny * 3] = f[i + nx*j + nx*ny * 3];
		ftemp[i + nx*jn + nx*ny * 4] = f[i + nx*j + nx*ny * 4];
		ftemp[in + nx*jn + nx*ny * 7] = f[i + nx*j + nx*ny * 7];
	}
	else if ((i == nx - 1) && (j == 0)) {						//BOTTOM-RIGHT

		in = i - 1;
		ip = i + 1;
		jn = j - 1;
		jp = j + 1;

		ftemp[i + nx*j + nx*ny * 0] = f[i + nx*j + nx*ny * 0];
		ftemp[i + nx*jp + nx*ny * 2] = f[i + nx*j + nx*ny * 2];
		ftemp[in + nx*j + nx*ny * 3] = f[i + nx*j + nx*ny * 3];
		ftemp[in + nx*jp + nx*ny * 6] = f[i + nx*j + nx*ny * 6];
	}
}
void LBM_GPU::Streaming() {

	dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
	dim3 dimGrid((nx + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, (ny + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y, (a + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z);
	Kernel_Streaming << < dimGrid, dimBlock >> > (d_f, d_ftemp, d_is_boundary_node, nx, ny, a);

}

__global__ 
void Kernel_BC_bounceback(float* f, float* ftemp, int nx, int ny, int a) {

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	int k = blockDim.z * blockIdx.z + threadIdx.z;
	if (i >= nx || j >= ny || k >= a) return;
	else if ((i == 0) && (j > 0 && j < ny - 1)){						//LEFT
		ftemp[i + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 3];
		ftemp[i + nx*j + nx*ny * 8] = f[i + nx*j + nx*ny * 6];
		ftemp[i + nx*j + nx*ny * 5] = f[i + nx*j + nx*ny * 7];
	}
	else if ((i == nx - 1) && (j > 0 && j < ny - 1)) {			//RIGHT
		ftemp[i + nx*j + nx*ny * 3] = f[i + nx*j + nx*ny * 1];
		ftemp[i + nx*j + nx*ny * 6] = f[i + nx*j + nx*ny * 8];
		ftemp[i + nx*j + nx*ny * 7] = f[i + nx*j + nx*ny * 5];
	}
	else if ((i > 0 && i < nx - 1) && (j == 0)) {				//BOTTOM
		ftemp[i + nx*j + nx*ny * 2] = f[i + nx*j + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 5] = f[i + nx*j + nx*ny * 7];
		ftemp[i + nx*j + nx*ny * 6] = f[i + nx*j + nx*ny * 8];
	}
	else if ((i == 0) && (j == 0)) {							//BOTTOM-LEFT
		ftemp[i + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 3];
		ftemp[i + nx*j + nx*ny * 2] = f[i + nx*j + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 5] = f[i + nx*j + nx*ny * 7];
		ftemp[i + nx*j + nx*ny * 6] = f[i + nx*j + nx*ny * 8];
		ftemp[i + nx*j + nx*ny * 8] = f[i + nx*j + nx*ny * 6];
	}
	else if ((i == nx - 1) && (j == 0)) {						//BOTTOM-RIGHT
		ftemp[i + nx*j + nx*ny * 2] = f[i + nx*j + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 6] = f[i + nx*j + nx*ny * 8];
		ftemp[i + nx*j + nx*ny * 3] = f[i + nx*j + nx*ny * 1];
		ftemp[i + nx*j + nx*ny * 5] = f[i + nx*j + nx*ny * 7];
		ftemp[i + nx*j + nx*ny * 7] = f[i + nx*j + nx*ny * 5];
	}
}
void LBM_GPU::BC_bounceback() {

	dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
	dim3 dimGrid((nx + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, (ny + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y, (a + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z);
	Kernel_BC_bounceback << < dimGrid, dimBlock >> > (d_f, d_ftemp, nx, ny, a);

}

__global__ 
void Kernel_BC_vel(float* ftemp, float* rho, float Ux0, int nx, int ny, int a) {

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	int k = blockDim.z * blockIdx.z + threadIdx.z;
	if (i >= nx || j >= ny || k >= a) return;

	float rho0, ru;

	if ((i > 0 && i < nx - 1) && (j == ny - 1)) {				//TOP
		rho0 = ftemp[i + nx*j + nx*ny * 0] + ftemp[i + nx*j + nx*ny * 1] + ftemp[i + nx*j + nx*ny * 3]
			+ 2.0*(ftemp[i + nx*j + nx*ny * 2] + ftemp[i + nx*j + nx*ny * 5] + ftemp[i + nx*j + nx*ny * 6]);
		ru = rho0 * Ux0;

		ftemp[i + nx*j + nx*ny * 4] = ftemp[i + nx*j + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 7] = ftemp[i + nx*j + nx*ny * 5] - (1.0 / 2.0)*ru + (1.0 / 2.0)*(ftemp[i + nx*j + nx*ny * 1] - ftemp[i + nx*j + nx*ny * 3]);
		ftemp[i + nx*j + nx*ny * 8] = ftemp[i + nx*j + nx*ny * 6] + (1.0 / 2.0)*ru - (1.0 / 2.0)*(ftemp[i + nx*j + nx*ny * 1] - ftemp[i + nx*j + nx*ny * 3]);
	}
	else if ((i == 0) && (j == ny - 1)) {							//TOP-LEFT
		ftemp[i + nx*j + nx*ny * 1] = ftemp[i + nx*j + nx*ny * 3];
		ftemp[i + nx*j + nx*ny * 4] = ftemp[i + nx*j + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 8] = ftemp[i + nx*j + nx*ny * 6];
		ftemp[i + nx*j + nx*ny * 5] = 0.5 * (rho[(i + 1) + nx*(j - 1)] - (ftemp[i + nx*j + nx*ny * 0] + ftemp[i + nx*j + nx*ny * 1] + ftemp[i + nx*j + nx*ny * 2]
			+ ftemp[i + nx*j + nx*ny * 3] + ftemp[i + nx*j + nx*ny * 4] + ftemp[i + nx*j + nx*ny * 6] + ftemp[i + nx*j + nx*ny * 8]));
		ftemp[i + nx*j + nx*ny * 7] = ftemp[i + nx*j + nx*ny * 5];
	}
	else if ((i == nx - 1) && (j == ny - 1)) {						//TOP-RIGHT
		ftemp[i + nx*j + nx*ny * 3] = ftemp[i + nx*j + nx*ny * 1];
		ftemp[i + nx*j + nx*ny * 4] = ftemp[i + nx*j + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 7] = ftemp[i + nx*j + nx*ny * 5];
		ftemp[i + nx*j + nx*ny * 6] = 0.5 * (rho[(i - 1) + nx*(j - 1)] - (ftemp[i + nx*j + nx*ny * 0] + ftemp[i + nx*j + nx*ny * 1] + ftemp[i + nx*j + nx*ny * 2]
			+ ftemp[i + nx*j + nx*ny * 3] + ftemp[i + nx*j + nx*ny * 4] + ftemp[i + nx*j + nx*ny * 5] + ftemp[i + nx*j + nx*ny * 7]));
		ftemp[i + nx*j + nx*ny * 8] = ftemp[i + nx*j + nx*ny * 6];
	}

}
void LBM_GPU::BC_vel() {

	dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
	dim3 dimGrid((nx + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, (ny + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y, (a + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z);
	Kernel_BC_vel << < dimGrid, dimBlock >> > (d_ftemp, d_rho, Ux0, nx, ny, a);
}

__global__ 
void Kernel_Eq(float* ftemp, float* feq, float* Ux, float* Uy, float* rho, float* ex, float* ey, int nx, int ny, int a) {

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	int k = blockDim.z * blockIdx.z + threadIdx.z;
	if (i >= nx || j >= ny || k >= a) return;


	//Calculation of Macroscopic var 
	rho[i + nx*j] = ftemp[i + nx*j + nx*ny * 0] + ftemp[i + nx*j + nx*ny * 1]
		+ ftemp[i + nx*j + nx*ny * 2] + ftemp[i + nx*j + nx*ny * 3] + ftemp[i + nx*j + nx*ny * 4]
		+ ftemp[i + nx*j + nx*ny * 5] + ftemp[i + nx*j + nx*ny * 6] + ftemp[i + nx*j + nx*ny * 7]
		+ ftemp[i + nx*j + nx*ny * 8];

	Ux[i + nx*j] = ftemp[i + nx*j + nx*ny * 1] * ex[1] + ftemp[i + nx*j + nx*ny * 3] * ex[3]
		+ ftemp[i + nx*j + nx*ny * 5] * ex[5] + ftemp[i + nx*j + nx*ny * 6] * ex[6] + ftemp[i + nx*j + nx*ny * 7] * ex[7]
		+ ftemp[i + nx*j + nx*ny * 8] * ex[8];

	Uy[i + nx*j] = ftemp[i + nx*j + nx*ny * 2] * ey[2] + ftemp[i + nx*j + nx*ny * 4] * ey[4]
		+ ftemp[i + nx*j + nx*ny * 5] * ey[5] + ftemp[i + nx*j + nx*ny * 6] * ey[6] + ftemp[i + nx*j + nx*ny * 7] * ey[7]
		+ ftemp[i + nx*j + nx*ny * 8] * ey[8];

	Ux[i + nx*j] /= rho[i + nx*j];
	Uy[i + nx*j] /= rho[i + nx*j];



	feq[i + nx*j + nx*ny * 0] = (4.0 / 9.0) * rho[i + nx*j] * (1.0 - 1.5*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
	feq[i + nx*j + nx*ny * 1] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 + 3.0 * Ux[i + nx*j] + 4.5*pow(Ux[i + nx*j], 2) - 1.5*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
	feq[i + nx*j + nx*ny * 2] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 + 3.0 * Uy[i + nx*j] + 4.5*pow(Uy[i + nx*j], 2) - 1.5*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
	feq[i + nx*j + nx*ny * 3] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 - 3.0 * Ux[i + nx*j] + 4.5*pow(Ux[i + nx*j], 2) - 1.5*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
	feq[i + nx*j + nx*ny * 4] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 - 3.0 * Uy[i + nx*j] + 4.5*pow(Uy[i + nx*j], 2) - 1.5*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
	feq[i + nx*j + nx*ny * 5] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + 3.0 * (Ux[i + nx*j] + Uy[i + nx*j]) + 4.5*pow(Ux[i + nx*j] + Uy[i + nx*j], 2) - 1.5*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
	feq[i + nx*j + nx*ny * 6] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + 3.0 * (-Ux[i + nx*j] + Uy[i + nx*j]) + 4.5*pow(-Ux[i + nx*j] + Uy[i + nx*j], 2) - 1.5*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
	feq[i + nx*j + nx*ny * 7] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + 3.0 * (-Ux[i + nx*j] - Uy[i + nx*j]) + 4.5*pow(-Ux[i + nx*j] - Uy[i + nx*j], 2) - 1.5*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
	feq[i + nx*j + nx*ny * 8] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + 3.0 * (Ux[i + nx*j] - Uy[i + nx*j]) + 4.5*pow(Ux[i + nx*j] - Uy[i + nx*j], 2) - 1.5*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));


}
__global__
void Kernel_Collision(float* fN, float* ftemp, float* feq, int nx, int ny, int a, float tau) {

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	int k = blockDim.z * blockIdx.z + threadIdx.z;
	if (i >= nx || j >= ny || k >= a) return;


	fN[i + nx*j + nx*ny*k] = ftemp[i + nx*j + nx*ny*k] - (ftemp[i + nx*j + nx*ny*k] - feq[i + nx*j + nx*ny*k]) / tau;

}
void LBM_GPU::Collision() {

	dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
	dim3 dimGrid((nx + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, (ny + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y, (a + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z);
	Kernel_Eq << < dimGrid, dimBlock >> > (d_ftemp, d_feq, d_Ux, d_Uy, d_rho, d_ex, d_ey, nx, ny, a);
	Kernel_Collision << < dimGrid, dimBlock >> > (d_fN, d_ftemp, d_feq, nx, ny, a, tau);
}

__global__ 
void Kernel_Error(float* ftemp, float* f, float* Ux, float* Uy, float* U, float* rho, float* fN, float* UxN, float* UyN, float* UN, float* rhoN, float* ex, float* ey, int nx, int ny, int a) {
	
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	int k = blockDim.z * blockIdx.z + threadIdx.z;
	if (i >= nx || j >= ny || k >= a) return;


	rho[i + nx*j] = f[i + nx*j + nx*ny * 0] + f[i + nx*j + nx*ny * 1]
		+ f[i + nx*j + nx*ny * 2] + f[i + nx*j + nx*ny * 3] + f[i + nx*j + nx*ny * 4]
		+ f[i + nx*j + nx*ny * 5] + f[i + nx*j + nx*ny * 6] + f[i + nx*j + nx*ny * 7]
		+ f[i + nx*j + nx*ny * 8];

	Ux[i + nx*j] = f[i + nx*j + nx*ny * 1] * ex[1] + f[i + nx*j + nx*ny * 3] * ex[3]
		+ f[i + nx*j + nx*ny * 5] * ex[5] + f[i + nx*j + nx*ny * 6] * ex[6] + f[i + nx*j + nx*ny * 7] * ex[7]
		+ f[i + nx*j + nx*ny * 8] * ex[8];

	Uy[i + nx*j] = f[i + nx*j + nx*ny * 2] * ey[2] + f[i + nx*j + nx*ny * 4] * ey[4]
		+ f[i + nx*j + nx*ny * 5] * ey[5] + f[i + nx*j + nx*ny * 6] * ey[6] + f[i + nx*j + nx*ny * 7] * ey[7]
		+ f[i + nx*j + nx*ny * 8] * ey[8];

	Ux[i + nx*j] /= rho[i + nx*j];
	Uy[i + nx*j] /= rho[i + nx*j];
	U[i + nx*j] = sqrt(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2));




	rhoN[i + nx*j] = fN[i + nx*j + nx*ny * 0] + fN[i + nx*j + nx*ny * 1]
		+ fN[i + nx*j + nx*ny * 2] + fN[i + nx*j + nx*ny * 3] + fN[i + nx*j + nx*ny * 4]
		+ fN[i + nx*j + nx*ny * 5] + fN[i + nx*j + nx*ny * 6] + fN[i + nx*j + nx*ny * 7]
		+ fN[i + nx*j + nx*ny * 8];

	UxN[i + nx*j] = fN[i + nx*j + nx*ny * 1] * ex[1] + fN[i + nx*j + nx*ny * 3] * ex[3]
		+ fN[i + nx*j + nx*ny * 5] * ex[5] + fN[i + nx*j + nx*ny * 6] * ex[6] + fN[i + nx*j + nx*ny * 7] * ex[7]
		+ fN[i + nx*j + nx*ny * 8] * ex[8];

	UyN[i + nx*j] = fN[i + nx*j + nx*ny * 2] * ey[2] + fN[i + nx*j + nx*ny * 4] * ey[4]
		+ fN[i + nx*j + nx*ny * 5] * ey[5] + fN[i + nx*j + nx*ny * 6] * ey[6] + fN[i + nx*j + nx*ny * 7] * ey[7]
		+ fN[i + nx*j + nx*ny * 8] * ey[8];

	UxN[i + nx*j] /= rhoN[i + nx*j];
	UyN[i + nx*j] /= rhoN[i + nx*j];
	UN[i + nx*j] = sqrt(pow(UxN[i + nx*j], 2) + pow(UyN[i + nx*j], 2));

}
void LBM_GPU::Error() {

	dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
	dim3 dimGrid((nx + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, (ny + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y, (a + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z);
	Kernel_Error << < dimGrid, dimBlock >> > (d_ftemp, d_f, d_Ux, d_Uy, d_U, d_rho, d_fN, d_UxN, d_UyN, d_UN, d_rhoN, d_ex, d_ey, nx, ny, a);

	hipMemcpy(U, d_U, nx*ny * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(UN, d_UN, nx*ny * sizeof(float), hipMemcpyDeviceToHost);
}

__global__ 
void Kernel_Update(float* fN, float* f, float* Ux, float* Uy, float* U, float* rho, float* ex, float* ey, int nx, int ny, int a) {

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	int k = blockDim.z * blockIdx.z + threadIdx.z;
	if (i >= nx || j >= ny || k >= a) return;

	f[i + nx*j + nx*ny*k] = fN[i + nx*j + nx*ny*k];

	rho[i + nx*j] = f[i + nx*j + nx*ny * 0] + f[i + nx*j + nx*ny * 1]
		+ f[i + nx*j + nx*ny * 2] + f[i + nx*j + nx*ny * 3] + f[i + nx*j + nx*ny * 4]
		+ f[i + nx*j + nx*ny * 5] + f[i + nx*j + nx*ny * 6] + f[i + nx*j + nx*ny * 7]
		+ f[i + nx*j + nx*ny * 8];

	Ux[i + nx*j] = f[i + nx*j + nx*ny * 1] * ex[1] + f[i + nx*j + nx*ny * 3] * ex[3]
		+ f[i + nx*j + nx*ny * 5] * ex[5] + f[i + nx*j + nx*ny * 6] * ex[6] + f[i + nx*j + nx*ny * 7] * ex[7]
		+ f[i + nx*j + nx*ny * 8] * ex[8];

	Uy[i + nx*j] = f[i + nx*j + nx*ny * 2] * ey[2] + f[i + nx*j + nx*ny * 4] * ey[4]
		+ f[i + nx*j + nx*ny * 5] * ey[5] + f[i + nx*j + nx*ny * 6] * ey[6] + f[i + nx*j + nx*ny * 7] * ey[7]
		+ f[i + nx*j + nx*ny * 8] * ey[8];

	Ux[i + nx*j] /= rho[i + nx*j];
	Uy[i + nx*j] /= rho[i + nx*j];
	U[i + nx*j] = sqrt(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2));
}
void LBM_GPU::Update() {


	dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
	dim3 dimGrid((nx + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, (ny + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y, (a + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z);
	Kernel_Update << < dimGrid, dimBlock >> > (d_fN, d_f, d_Ux, d_Uy, d_U, d_rho, d_ex, d_ey, nx, ny, a);

}

void LBM_GPU::Print() {

	hipMemcpy(Ux, d_Ux, nx*ny * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(Uy, d_Uy, nx*ny * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(U, d_U, nx*ny * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(rho, d_rho, nx*ny * sizeof(float), hipMemcpyDeviceToHost);


// ============================================================================ //
//  VORTICITY
// ============================================================================ //
	//INNER
	for (i = 1; i < nx - 1; i++) {
		for (j = 1; j < ny - 1; j++) {
			W[i + nx*j] = (Uy[(i + 1) + nx*j] - Uy[(i - 1) + nx*j]) / (2.0*del_y) - (Ux[i + nx*(j + 1)] - Ux[i + nx*(j - 1)]) / (2.0*del_y);
		}
	}

	//LEFT BOUNDARY
	i = 0;
	for (j = 1; j < ny - 1; j++) {
		W[i + nx*j] = (Uy[(i + 1) + nx*j] - 0.0) / (del_y)-(Ux[i + nx*(j + 1)] - Ux[i + nx*(j - 1)]) / (2.0*del_y);
	}

	//RIGHT BOUNDARY
	i = nx - 1;
	for (j = 1; j < ny - 1; j++) {
		W[i + nx*j] = (0.0 - Uy[(i - 1) + nx*j]) / (del_y)-(Ux[i + nx*(j + 1)] - Ux[i + nx*(j - 1)]) / (2.0*del_y);
	}

	//TOP BOUNDARY
	j = ny - 1;
	for (i = 1; i < nx - 1; i++) {
		W[i + nx*j] = (Uy[(i + 1) + nx*j] - Uy[(i - 1) + nx*j]) / (2.0*del_y) - (0.0 - Ux[i + nx*(j - 1)]) / (del_y);
	}

	//BOTTOM BOUNDARY
	j = 0;
	for (i = 1; i < nx - 1; i++) {
		W[i + nx*j] = (Uy[(i + 1) + nx*j] - Uy[(i - 1) + nx*j]) / (2.0*del_y) - (Ux[i + nx*(j + 1)] - 0.0) / (del_y);
	}

	//TOP-LEFT CONNER
	i = 0;
	j = ny - 1;
	W[i + nx*j] = (Uy[(i + 1) + nx*j] - 0.0) / (del_y)-(0.0 - Ux[i + nx*(j - 1)]) / (del_y);

	//BOTTOM-LEFT CONNER
	i = 0;
	j = 0;
	W[i + nx*j] = (Uy[(i + 1) + nx*j] - 0.0) / (del_y)-(Ux[i + nx*(j + 1)] - 0.0) / (del_y);

	//TOP-RIGHT CONNER
	i = nx - 1;
	j = ny - 1;
	W[i + nx*j] = (0.0 - Uy[(i - 1) + nx*j]) / (del_y)-(0.0 - Ux[i + nx*(j - 1)]) / (del_y);

	//BOTTOM-RIGHT CONNER
	i = nx - 1;
	j = 0;
	W[i + nx*j] = (0.0 - Uy[(i - 1) + nx*j]) / (del_y)-(Ux[i + nx*(j + 1)] - 0.0) / (del_y);
// ============================================================================ //



// ============================================================================ //
//  NORMALIZATION 
// ============================================================================ //
	for (i = 0; i < nx; i++) {
		for (j = 0; j < ny; j++) {
			Ux_p[i + nx*j] = Ux[i + nx*j] / Ux0;
			Uy_p[i + nx*j] = Uy[i + nx*j] / Ux0;
			U_p[i + nx*j] = U[i + nx*j] / Ux0;
			W[i + nx*j] = W[i + nx*j] / Ux0;
		}
	}
// ============================================================================ //






	fout_GPU << endl;
	fout_GPU << "variables = X Y Ux Uy U rho W" << endl;
	fout_GPU << "zone i=" << nx << " j=" << ny << endl;
	for (j = 0; j < ny; j++) {
		for (i = 0; i < nx; i++) {
			fout_GPU << i << "\t" << j << "\t" << Ux_p[i + nx*j] << "\t" << Uy_p[i + nx*j] << "\t"
				<< U_p[i + nx*j] << "\t" << rho[i + nx*j] << "\t" << W[i + nx*j] << endl;
		}
	}

	fout_GPU_Ux << "variables = X Y Ux " << endl;
	i = nx / 2;
	for (j = 0; j < ny; j++) {
		fout_GPU_Ux << i << "\t" << j << "\t" << Ux_p[i + nx*j] << endl;
	}

	fout_GPU_Uy << "variables = X Y Uy " << endl;
	j = ny / 2;
	for (i = 0; i < nx; i++) {
		fout_GPU_Uy << i << "\t" << j << "\t" << Uy_p[i + nx*j] << endl;
	}

}

LBM_GPU::~LBM_GPU()
{
	hipFree(d_is_boundary_node);
	hipFree(d_f);
	hipFree(d_fN);
	hipFree(d_ftemp);
	hipFree(d_feq);
	hipFree(d_Ux);
	hipFree(d_Uy);
	hipFree(d_rho);
	hipFree(d_ex);
	hipFree(d_ey);
	hipFree(d_U);
	hipFree(d_UN);
	hipFree(d_UxN);
	hipFree(d_UyN);
	hipFree(rhoN);


	delete[] Uy_p;
	delete[] Ux_p;
	delete[] U_p;
	delete[] ey;
	delete[] ex;
	delete[] fN;
	delete[] feq;
	delete[] ftemp;
	delete[] f;
	delete[] rhoN;
	delete[] UyN;
	delete[] UxN;
	delete[] UN;
	delete[] W;
	delete[] rho;
	delete[] Uy;
	delete[] Ux;
	delete[] U;
	delete[] is_boundary_node;
	cout << endl << "Done!" << endl;
}
